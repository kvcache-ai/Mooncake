#include "hip/hip_runtime.h"
#include <mooncake_backend.h>
#include <mooncake_worker.cuh>

namespace mooncake {

class MooncakeWork : public ::c10d::Work {
   public:
    MooncakeWork(c10d::OpType opType, hipEvent_t event)
        : Work(-1, opType), event_(event) {}

    bool isCompleted() override {
        return hipEventQuery(event_) == hipSuccess;
    }

    bool wait(std::chrono::milliseconds timeout) override {
        return hipEventSynchronize(event_) == hipSuccess;
    }

   private:
    hipEvent_t event_;
};

__device__ int findIdleTask(Task* tasks, size_t numTasks) {
    for (size_t i = 0; i < numTasks; ++i) {
        int expected = IDLE;
        if (atomicCAS((int*)&tasks[i].status, expected, OCCUPIED) == expected) {
            return i;
        }
    }
    return -1;
}

__global__ void enqueueTaskKernel(c10d::OpType opType, size_t tensorSize,
                                  Task* tasks, size_t numTasks) {
    // Find idle task
    int idx = findIdleTask(tasks, numTasks);
    assert(idx >= 0);

    // Copy task into slot
    tasks[idx].opType = opType;
    tasks[idx].tensorSize = tensorSize;

    // Mark READY
    __threadfence();  // Ensure writes visible to host
    tasks[idx].status = READY;

    // Spin-wait until CPU proxy sets DONE
    while (atomicAdd((int*)&tasks[idx].status, 0) != DONE) {
        __threadfence();
    }
    tasks[idx].status = IDLE;
}

MooncakeWorker::MooncakeWorker(TransferEngine* engine, int rank, int size)
    : engine_(engine), rank_(rank), size_(size) {
    // Pin memory for task array
    hipHostAlloc(&tasks_, kNumTasks_ * sizeof(Task), hipHostMallocMapped);
    hipHostGetDevicePointer(&tasks_device_, tasks_, 0);
    for (size_t i = 0; i < kNumTasks_; ++i) {
        tasks_[i].status = IDLE;
    }
}

c10::intrusive_ptr<c10d::Work> MooncakeWorker::putTask(
    c10d::OpType opType, size_t tensorSize, hipStream_t stream,
    const std::function<void(void* dst)>& tensorToBuffer,
    const std::function<void(void* src)>& bufferToTensor) {
    tensorToBuffer((void*)segment_descs_[rank_]->buffers[0].addr);
    enqueueTaskKernel<<<1, 1, 0, stream>>>(opType, tensorSize, tasks_device_,
                                           kNumTasks_);
    bufferToTensor((void*)segment_descs_[rank_]->buffers[1].addr);
    hipEvent_t event;
    hipEventCreateWithFlags(&event, hipEventDisableTiming);
    hipEventRecord(event, stream);
    return c10::make_intrusive<MooncakeWork>(opType, event);
}

}  // namespace mooncake